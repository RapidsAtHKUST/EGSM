#include <algorithm>
#include <cstdint>
#include <fstream>
#include <unordered_map>

#include "utils/cuda_helpers.h"
#include "graph/graph.h"
#include "graph/graph_gpu.h"


GraphGPU::GraphGPU(const Graph& g)
: vlabels_(nullptr)
, offsets_(nullptr)
, neighbors_(nullptr)
{
    cudaErrorCheck(hipMalloc(&vlabels_, sizeof(uint32_t) * g.vcount_));
    cudaErrorCheck(hipMalloc(&offsets_, sizeof(uint32_t) * (g.vcount_ + 1)));
    cudaErrorCheck(hipMalloc(&neighbors_, sizeof(uint32_t) * (g.ecount_ * 2 + 1)));
    cudaErrorCheck(hipMemcpy(vlabels_, g.vlabels_, sizeof(uint32_t) * g.vcount_, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(offsets_, g.offsets_, sizeof(uint32_t) * (g.vcount_ + 1), hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(neighbors_, g.neighbors_, sizeof(uint32_t) * (g.ecount_ * 2 + 1), hipMemcpyHostToDevice));
}


void GraphGPU::Deallocate()
{
    cudaErrorCheck(hipFree(vlabels_));
    cudaErrorCheck(hipFree(offsets_));
    cudaErrorCheck(hipFree(neighbors_));
}
